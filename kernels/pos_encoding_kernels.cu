#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>

#include "cuda_compat.h"
#include "dispatch_utils.h"

namespace aphrodite {

template <typename scalar_t, typename input_type, bool IS_NEOX,
            bool use_dequant>
  inline __device__ void apply_rotary_embedding(
      input_type * __restrict__ arr, const scalar_t *__restrict__ cos_ptr,
      const scalar_t *__restrict__ sin_ptr, int rot_offset, int embed_dim,
      scalar_t *__restrict__ arr_out = nullptr, const float scale = 1.0f) {
    int x_index, y_index;
    scalar_t cos, sin;
    if constexpr (IS_NEOX) {
      // GPT-NeoX style rotary embedding.
      x_index = rot_offset;
      y_index = embed_dim + rot_offset;
      cos = __ldg(cos_ptr + x_index);
      sin = __ldg(sin_ptr + x_index);
    } else {
      // GPT-J style rotary embedding.
      x_index = 2 * rot_offset;
      y_index = 2 * rot_offset + 1;
      cos = __ldg(cos_ptr + x_index / 2);
      sin = __ldg(sin_ptr + x_index / 2);
    }
    if constexpr (use_dequant) {
      const scalar_t x = (scalar_t)((float)arr[x_index] * scale);
      const scalar_t y = (scalar_t)((float)arr[y_index] * scale);
      arr_out[x_index] = x * cos - y * sin;
      arr_out[y_index] = y * cos + x * sin;
    } else {
      const scalar_t x = arr[x_index];
      const scalar_t y = arr[y_index];
      arr[x_index] = x * cos - y * sin;
      arr[y_index] = y * cos + x * sin;
    }
  }

template <typename scalar_t, typename input_type, bool IS_NEOX,
            bool use_dequant>
  __global__ void rotary_embedding_kernel(
      const int64_t *__restrict__ positions, 
      input_type *__restrict__ query, 
      input_type *__restrict__ key,
      const scalar_t *__restrict__ cos_sin_cache,
      const int rot_dim, const int query_stride, const int key_stride,
      const int num_heads, const int num_kv_heads, const int head_size,
      scalar_t *__restrict__ query_out = nullptr,
      scalar_t * __restrict__ key_out = nullptr,
      const int query_out_stride = 1,
      const int key_out_stride = 1,
      const float query_scale = 1.0f,
      const float key_scale = 1.0f) {
    // Each thread block is responsible for one token.
    const int token_idx = blockIdx.x;
    int64_t pos = positions[token_idx];
    const scalar_t *cache_ptr = cos_sin_cache + pos * rot_dim;

    const int embed_dim = rot_dim / 2;
    const scalar_t *cos_ptr = cache_ptr;
    const scalar_t *sin_ptr = cache_ptr + embed_dim;

    const int nq = num_heads * embed_dim;
    for (int i = threadIdx.x; i < nq; i += blockDim.x) {
      const int head_idx = i / embed_dim;
      const int token_head = token_idx * query_stride + head_idx * head_size;
      const int rot_offset = i % embed_dim;
      if constexpr (use_dequant) {
        const int token_out_head =
            token_idx * query_out_stride + head_idx * head_size;
        apply_rotary_embedding<scalar_t, input_type, IS_NEOX, use_dequant>(
            query + token_head, cos_ptr, sin_ptr, rot_offset, embed_dim,
            query_out + token_out_head, query_scale);
      } else {
        apply_rotary_embedding<scalar_t, input_type, IS_NEOX, use_dequant>(
            query + token_head, cos_ptr, sin_ptr, rot_offset, embed_dim);
      }
    }

    const int nk = num_kv_heads * embed_dim;
    for (int i = threadIdx.x; i < nk; i += blockDim.x) {
      const int head_idx = i / embed_dim;
      const int token_head = token_idx * key_stride + head_idx * head_size;
      const int rot_offset = i % embed_dim;
      if (use_dequant) {
        const int token_out_head =
            token_idx * key_out_stride + head_idx * head_size;
        apply_rotary_embedding<scalar_t, input_type, IS_NEOX, use_dequant>(
            key + token_head, cos_ptr, sin_ptr, rot_offset, embed_dim,
            key_out + token_out_head, key_scale);
      } else {
        apply_rotary_embedding<scalar_t, input_type, IS_NEOX, use_dequant>(
            key + token_head, cos_ptr, sin_ptr, rot_offset, embed_dim);
      }
    }
}

} // namespace aphrodite

void rotary_embedding(
    torch::Tensor &positions, // [batch_size, seq_len] or [num_tokens]
    torch::Tensor &query,     // [batch_size, seq_len, num_heads * head_size] or [num_tokens, num_heads * head_size]
    torch::Tensor &key,       // [batch_size, seq_len, num_heads * head_size] or [num_tokens, num_heads * head_size]
    int head_size,
    torch::Tensor &cos_sin_cache, // [max_position, rot_dim]
    bool is_neox,
    torch::Tensor &query_out, // [batch_size, seq_len, num_heads * head_size] or [num_tokens, num_heads * head_size]
    torch::Tensor &key_out, // [batch_size, seq_len, num_heads * head_size] or [num_tokens, num_heads * head_size]
    bool use_dequant = false,
    float query_scale = 1.0f,
    float key_scale = 1.0f) {
  int64_t num_tokens = query.numel() / query.size(-1);
  int rot_dim = cos_sin_cache.size(1);
  int num_heads = query.size(-1) / head_size;
  int num_kv_heads = key.size(-1) / head_size;
  int64_t query_stride = query.stride(-2);
  int64_t key_stride = key.stride(-2);

  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * rot_dim / 2, 512));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(query));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  APHRODITE_DISPATCH_FLOATING_TYPES(
    cos_sin_cache.scalar_type(), "rotary_embedding_kernel", [&] {
        if (use_dequant) {
          int query_out_stride = query_out.stride(-2);
          int key_out_stride = key_out.stride(-2);
          if (is_neox) {
            aphrodite::rotary_embedding_kernel<scalar_t, int32_t, true, true>
                <<<grid, block, 0, stream>>>(
                    positions.data_ptr<int64_t>(), query.data_ptr<int32_t>(),
                    key.data_ptr<int32_t>(), cos_sin_cache.data_ptr<scalar_t>(),
                    rot_dim, query_stride, key_stride, num_heads, num_kv_heads,
                    head_size, query_out.data_ptr<scalar_t>(),
                    key_out.data_ptr<scalar_t>(), query_out_stride,
                    key_out_stride, query_scale, key_scale);
          } else {
            aphrodite::rotary_embedding_kernel<scalar_t, int32_t, false, true>
                <<<grid, block, 0, stream>>>(
                    positions.data_ptr<int64_t>(), query.data_ptr<int32_t>(),
                    key.data_ptr<int32_t>(), cos_sin_cache.data_ptr<scalar_t>(),
                    rot_dim, query_stride, key_stride, num_heads, num_kv_heads,
                    head_size, query_out.data_ptr<scalar_t>(),
                    key_out.data_ptr<scalar_t>(), query_out_stride,
                    key_out_stride, query_scale, key_scale);
          }
        } else {
          if (is_neox) {
            aphrodite::rotary_embedding_kernel<scalar_t, scalar_t, true, false>
                <<<grid, block, 0, stream>>>(
                    positions.data_ptr<int64_t>(), query.data_ptr<scalar_t>(),
                    key.data_ptr<scalar_t>(),
                    cos_sin_cache.data_ptr<scalar_t>(), rot_dim, query_stride,
                    key_stride, num_heads, num_kv_heads, head_size);
          } else {
            aphrodite::rotary_embedding_kernel<scalar_t, scalar_t, false, false>
                <<<grid, block, 0, stream>>>(
                    positions.data_ptr<int64_t>(), query.data_ptr<scalar_t>(),
                    key.data_ptr<scalar_t>(),
                    cos_sin_cache.data_ptr<scalar_t>(), rot_dim, query_stride,
                    key_stride, num_heads, num_kv_heads, head_size);
          }
        }
      });
}
