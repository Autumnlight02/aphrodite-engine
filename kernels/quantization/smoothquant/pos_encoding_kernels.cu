#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "../../cuda_compat.h"
#include "../../dispatch_utils.h"

namespace aphrodite {
template <typename scalar_t, bool IS_NEOX>
inline __device__ void apply_dequant_rotary_embedding(
  int32_t* __restrict__ arr,
  const scalar_t* __restrict__ cos_ptr,
  const scalar_t* __restrict__ sin_ptr,
  int rot_offset,
  int embed_dim,
  scalar_t* __restrict__ arr_out,
  const float scale) {
  int x_index, y_index;
  scalar_t cos, sin;
  if constexpr (IS_NEOX) {
    // GPT-NeoX style rotary embedding.
    x_index = rot_offset;
    y_index = embed_dim + rot_offset;
    cos = APHRODITE_LDG(cos_ptr + x_index);
    sin = APHRODITE_LDG(sin_ptr + x_index);
  } else {
    // GPT-J style rotary embedding.
    x_index = 2 * rot_offset;
    y_index = 2 * rot_offset + 1;
    cos = APHRODITE_LDG(cos_ptr + x_index / 2);
    sin = APHRODITE_LDG(sin_ptr + x_index / 2);
  }
  const scalar_t x = (scalar_t)((float)arr[x_index] * scale);
  const scalar_t y = (scalar_t)((float)arr[y_index] * scale);
  arr_out[x_index] = x * cos - y * sin;
  arr_out[y_index] = y * cos + x * sin;
}

template <typename scalar_t, bool IS_NEOX>
__global__ void dequant_rotary_embedding_kernel(
  const int64_t* __restrict__ positions, 
  int32_t* __restrict__ query, 
  int32_t* __restrict__ key,
  const scalar_t* __restrict__ cos_sin_cache,
  const int rot_dim,
  const int64_t query_stride,
  const int64_t key_stride,
  const int num_heads,
  const int num_kv_heads,
  const int head_size,
  scalar_t* __restrict__ query_out,
  scalar_t* __restrict__ key_out,
  const int query_out_stride,
  const int key_out_stride,
  const float query_scale,
  const float key_scale) {
  // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const scalar_t *cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const scalar_t *cos_ptr = cache_ptr;
  const scalar_t *sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    const int token_out_head =
        token_idx * query_out_stride + head_idx * head_size;
    apply_dequant_rotary_embedding<scalar_t, IS_NEOX>(
        query + token_head, cos_ptr, sin_ptr, rot_offset, embed_dim,
        query_out + token_out_head, query_scale);
  }

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;
    const int rot_offset = i % embed_dim;
    const int token_out_head =
        token_idx * key_out_stride + head_idx * head_size;
    apply_dequant_rotary_embedding<scalar_t, IS_NEOX>(
        key + token_head, cos_ptr, sin_ptr, rot_offset, embed_dim,
        key_out + token_out_head, key_scale);
  }
}
} // namespace aphrodite

void dequant_rotary_embedding(
  torch::Tensor& positions, // [batch_size, seq_len] or [num_tokens]
  torch::Tensor& query,     // [batch_size, seq_len, num_heads * head_size] or [num_tokens, num_heads * head_size]
  torch::Tensor& key,       // [batch_size, seq_len, num_heads * head_size] or [num_tokens, num_heads * head_size]
  int head_size,
  torch::Tensor& cos_sin_cache, // [max_position, rot_dim]
  bool is_neox,
  torch::Tensor& query_out, // [batch_size, seq_len, num_heads * head_size] or [num_tokens, num_heads * head_size]
  torch::Tensor& key_out, // [batch_size, seq_len, num_heads * head_size] or [num_tokens, num_heads * head_size]
  float query_scale,
  float key_scale) {
  int64_t num_tokens = query.numel() / query.size(-1);
  int rot_dim = cos_sin_cache.size(1);
  int num_heads = query.size(-1) / head_size;
  int num_kv_heads = key.size(-1) / head_size;
  int64_t query_stride = query.stride(-2);
  int64_t key_stride = key.stride(-2);

  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * rot_dim / 2, 512));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(query));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  APHRODITE_DISPATCH_FLOATING_TYPES(
      cos_sin_cache.scalar_type(), "dequant_rotary_embedding_kernel", [&] {
        int query_out_stride = query_out.stride(-2);
        int key_out_stride = key_out.stride(-2);
        if (is_neox) {
          aphrodite::dequant_rotary_embedding_kernel<scalar_t, true>
              <<<grid, block, 0, stream>>>(
                  positions.data_ptr<int64_t>(),
                  query.data_ptr<int32_t>(),
                  key.data_ptr<int32_t>(),
                  cos_sin_cache.data_ptr<scalar_t>(),
                  rot_dim,
                  query_stride,
                  key_stride,
                  num_heads,
                  num_kv_heads,
                  head_size,
                  query_out.data_ptr<scalar_t>(),
                  key_out.data_ptr<scalar_t>(),
                  query_out_stride,
                  key_out_stride,
                  query_scale,
                  key_scale);
        } else {
          aphrodite::dequant_rotary_embedding_kernel<scalar_t, false>
              <<<grid, block, 0, stream>>>(
                  positions.data_ptr<int64_t>(),
                  query.data_ptr<int32_t>(),
                  key.data_ptr<int32_t>(),
                  cos_sin_cache.data_ptr<scalar_t>(),
                  rot_dim,
                  query_stride,
                  key_stride,
                  num_heads,
                  num_kv_heads,
                  head_size,
                  query_out.data_ptr<scalar_t>(),
                  key_out.data_ptr<scalar_t>(),
                  query_out_stride,
                  key_out_stride,
                  query_scale,
                  key_scale);
          }
      });
}
