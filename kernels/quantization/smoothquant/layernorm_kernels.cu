#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>

#include "../../dispatch_utils.h"
#include "../../reduction_utils.cuh"
#include "quant_utils.cuh"

namespace aphrodite {

// TODO: Further optimize this kernel.
template <typename scalar_t>
__global__ void rms_norm_quant_kernel(
  int8_t* __restrict__ out,         // [..., hidden_size]
  const scalar_t* __restrict__ input, // [..., hidden_size]
  const scalar_t* __restrict__ weight, // [hidden_size]
  const float epsilon,
  const int num_tokens,
  const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    const float x = (float)input[blockIdx.x * hidden_size + idx];
    variance += x * x;
  }
  variance = blockReduceSum<float>(variance);
  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float)input[blockIdx.x * hidden_size + idx];
    out[blockIdx.x * hidden_size + idx] = float_to_int8_rn(
      x * s_variance * (float)(weight[idx]));
  }
}

template <typename scalar_t, bool use_per_token_dequant>
__global__ void dequant_add_residual_rms_norm_quant_kernel(
  const int32_t* __restrict__ input,
  scalar_t* __restrict__ residual,
  int8_t* __restrict__ out,
  const scalar_t* __restrict__ gamma,
  const float layernorm_eps,
  const float scale,
  const int num_tokens,
  const int hidden_size,
  const float* __restrict__ act_scale = nullptr) {
  // layernorm module in the T5 style No bias and no subtraction of mean.
  const int tid = threadIdx.x;

  __shared__ float s_variance;
  float variance = 0.0f;
  float local_var_sum = 0.0f;
  float scale_ = scale;
  if constexpr (use_per_token_dequant) {
    scale_ = scale * act_scale[blockIdx.x];
  }
  for (int i = tid; i < hidden_size; i += blockDim.x) {
    float diff = ((((float)input[blockIdx.x * hidden_size + i]) * scale_) +
                  (float)residual[blockIdx.x * hidden_size + i]);
    residual[blockIdx.x * hidden_size + i] = (scalar_t)diff;
    local_var_sum += diff * diff;
  }
  variance = blockReduceSum<float>(local_var_sum);

  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / (float)hidden_size + layernorm_eps);
  }
  __syncthreads();

  for (int i = tid; i < hidden_size; i += blockDim.x) {
    float x = (float) residual[blockIdx.x * hidden_size + i];
    out[blockIdx.x * hidden_size + i] = float_to_int8_rn(
        x * s_variance * (float)(gamma[i]));
  }
}

template<typename scalar_t>
__global__ void add_residual_rms_norm_quant_kernel(
  int8_t* __restrict__ out,             // [..., hidden_size]
  scalar_t* __restrict__ input,           // [..., hidden_size]
  scalar_t* __restrict__ residual,        // [..., hidden_size]
  const scalar_t* __restrict__ weight,    // [hidden_size]
  const float epsilon,
  const int num_tokens,
  const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float) input[blockIdx.x * hidden_size + idx];
    x += (float) residual[blockIdx.x * hidden_size + idx];
    variance += x * x;
    residual[blockIdx.x * hidden_size + idx] = (scalar_t) x;
  }
  variance = blockReduceSum<float>(variance);
  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float) residual[blockIdx.x * hidden_size + idx];
    out[blockIdx.x * hidden_size + idx] = float_to_int8_rn(x * s_variance * (float)(weight[idx]));
  }
}

} // namespace aphrodite

void rms_norm_quant(
  torch::Tensor& out,    // [..., hidden_size]
  torch::Tensor& input,  // [..., hidden_size]
  torch::Tensor& weight, // [hidden_size]
  float epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;
  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  APHRODITE_DISPATCH_FLOATING_TYPES(input.scalar_type(), "rms_norm_quant_kernel", [&] {
    aphrodite::rms_norm_quant_kernel<scalar_t><<<grid, block, 0, stream>>>(
      out.data_ptr<int8_t>(),
      input.data_ptr<scalar_t>(),
      weight.data_ptr<scalar_t>(),
      epsilon,
      num_tokens,
      hidden_size);
  });
}

void dequant_add_residual_rms_norm_quant(
  torch::Tensor& out,      // [..., hidden_size]
  torch::Tensor& input,    // [..., hidden_size]
  torch::Tensor& residual, // [..., hidden_size]
  torch::Tensor& gamma,    // [hidden_size]
  float scale,
  float epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;
  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  APHRODITE_DISPATCH_FLOATING_TYPES(
      residual.scalar_type(), "dequant_add_residual_rms_norm_quant_kernel",
      [&] {
          aphrodite::dequant_add_residual_rms_norm_quant_kernel<scalar_t, false><<<grid, block, 0, stream>>>(
            input.data_ptr<int32_t>(),
            residual.data_ptr<scalar_t>(),
            out.data_ptr<int8_t>(),
            gamma.data_ptr<scalar_t>(),
            epsilon,
            scale,
            num_tokens,
            hidden_size);
      });
}

void dequant_add_residual_rms_norm_quant(
  torch::Tensor& out,      // [..., hidden_size]
  torch::Tensor& input,    // [..., hidden_size]
  torch::Tensor& residual, // [..., hidden_size]
  torch::Tensor& gamma,    // [hidden_size]
  torch::Tensor& scale,    // [num_tokens]
  float epsilon,
  float weight_dequant_scale) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;

  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  APHRODITE_DISPATCH_FLOATING_TYPES(
      residual.scalar_type(), "dequant_add_residual_rms_norm_quant_kernel",
      [&] {
          aphrodite::dequant_add_residual_rms_norm_quant_kernel<scalar_t, true><<<grid, block, 0, stream>>>(
            input.data_ptr<int32_t>(),
            residual.data_ptr<scalar_t>(),
            out.data_ptr<int8_t>(),
            gamma.data_ptr<scalar_t>(),
            epsilon,
            weight_dequant_scale,
            num_tokens,
            hidden_size,
            scale.data_ptr<float>());
      });
}

void add_residual_rms_norm_quant(
  torch::Tensor& out,      // [..., hidden_size]
  torch::Tensor& input,    // [..., hidden_size]
  torch::Tensor& residual, // [..., hidden_size]
  torch::Tensor& weight,   // [hidden_size]
  float epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;

  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  APHRODITE_DISPATCH_FLOATING_TYPES(
    input.scalar_type(),
    "add_residual_rms_norm_quant_kernel", [&] {
      aphrodite::add_residual_rms_norm_quant_kernel<scalar_t><<<grid, block, 0, stream>>>(
        out.data_ptr<int8_t>(),
        input.data_ptr<scalar_t>(),
        residual.data_ptr<scalar_t>(),
        weight.data_ptr<scalar_t>(),
        epsilon,
        num_tokens,
        hidden_size);
    });
}
